#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define RANDOM(x) (rand() % x)

#define MAX 100000

#define BLOCKSIZE 16

__global__ void multiply(const int *a, const int *b, int *c, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    int sum = 0;

    if (row < n && col < n) {
        for (k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }

        c[row * n + col] = sum;
    }
}

int main(int argc, char **argv) {
    int n = 512;
    int i, j, k;
    timeval start, finish;

    if (argc == 2) {
        n = atoi(argv[1]);
    }

    int *host_a = (int *)malloc(sizeof(int) * n * n);
    int *host_b = (int *)malloc(sizeof(int) * n * n);
    int *host_c = (int *)malloc(sizeof(int) * n * n);
    int *host_c2 = (int *)malloc(sizeof(int) * n * n);

    srand(time(NULL));

    for (i = 0; i < n * n; i++) {
        host_a[i] = RANDOM(MAX);
        host_b[i] = RANDOM(MAX);
    }

    hipError_t error = hipSuccess;

    int *device_a, *device_b, *device_c;
    error = hipMalloc((void **)&device_a, sizeof(int) * n * n);
    error = hipMalloc((void **)&device_b, sizeof(int) * n * n);
    error = hipMalloc((void **)&device_c, sizeof(int) * n * n);

    if (error != hipSuccess) {
        printf("Fail to hipMalloc on GPU");
        return 1;
    }

//GPU parallel start
    gettimeofday(&start, 0);

    hipMemcpy(device_a, host_a, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, sizeof(int) * n * n, hipMemcpyHostToDevice);

    double num = ceil(pow((double)n,2) / pow((double)BLOCKSIZE, 2));
    int gridsize = (int)ceil(sqrt(num));

    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 dimGrid(gridsize, gridsize, 1);

    multiply<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, n);
    hipDeviceSynchronize();

    hipMemcpy(host_c, device_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    gettimeofday(&finish, 0);

    double t = 1000000 * (finish.tv_sec - start.tv_sec) + finish.tv_usec - start.tv_usec;
    printf("GPU: %lf ms\n", t / 1000);
//GPU parallel finish


//CPU serial start
    gettimeofday(&start, 0);

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            host_c2[i * n + j] = 0;
            for (k = 0; k < n; k++) {
                host_c2[i * n + j] += host_a[i * n + k] * host_b[k * n + j];
            }
        }
    }

    gettimeofday(&finish, 0);

    t = 1000000 * (finish.tv_sec - start.tv_sec) + finish.tv_usec - start.tv_usec;
    printf("CPU: %lf ms\n", t / 1000);
//CPU serial start

//check
    int errorNum = 0;
    for (int i = 0; i < n * n; i++) {
        if (host_c[i] != host_c2[i]) {
            errorNum ++;
            printf("Error occurs at index: %d: c = %d, c2 = %d\n", i, host_c[i], host_c2[i]);
        }
    }
    if (errorNum == 0) {
        printf("Successfully run on GPU and CPU!\n");
    } else {
        printf("%d error(s) occurs!\n", errorNum);
    }

    free(host_a);
    free(host_b);
    free(host_c);
    free(host_c2);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
