#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define RANDOM(x) (rand() % x)

#define MAX 100000

#define BLOCKSIZE 16


//主机调用，设备执行，返回为空
__global__ void add(const int *a, const int *b, int *c, int n) {
    int i = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}


int main(int argc, char **argv) {
    int n = 512;
    int i;
    timeval start, finish;

    if (argc == 2) {
        n = atoi(argv[1]);
    }
	
	//主机内存分配
    int *host_a = (int *)malloc(sizeof(int) * n);
    int *host_b = (int *)malloc(sizeof(int) * n);
    int *host_c = (int *)malloc(sizeof(int) * n);
    int *host_d = (int *)malloc(sizeof(int) * n);
	
	//数据初始匿
    srand(time(NULL));
    for (i = 0; i < n; i++) {
        host_a[i] = RANDOM(MAX);
        host_b[i] = RANDOM(MAX);
    }
	
	//设备内存分配
    hipError_t error = hipSuccess;
    int *device_a, *device_b, *device_c;
    error = hipMalloc((void **)&device_a, sizeof(int) * n);
    error = hipMalloc((void **)&device_b, sizeof(int) * n);
    error = hipMalloc((void **)&device_c, sizeof(int) * n);
	
    if (error != hipSuccess) {
        printf("Fail to hipMalloc on GPU");
        return 1;
    }

//GPU parallel start
    gettimeofday(&start, 0);
	//主机向设备拷贝数捿
    hipMemcpy(device_a, host_a, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, sizeof(int) * n, hipMemcpyHostToDevice);
	
    int gridsize = (int)ceil(sqrt(ceil(n / (BLOCKSIZE * BLOCKSIZE))));

    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);//设置参数
    dim3 dimGrid(gridsize, gridsize, 1);

    add<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, n);//启动kernel
    hipDeviceSynchronize();

    hipMemcpy(host_c, device_c, sizeof(int) * n, hipMemcpyDeviceToHost);//结果拷贝到host

    gettimeofday(&finish, 0);

    double t = 1000000 * (finish.tv_sec - start.tv_sec) + finish.tv_usec - start.tv_usec;
    printf("GPU: %lf ms\n", t / 1000);
//GPU parallel finish


//CPU serial start
    gettimeofday(&start, 0);

    for (i = 0; i < n; i++) {
        host_d[i] = host_a[i] + host_b[i];
    }

    gettimeofday(&finish, 0);

    t = 1000000 * (finish.tv_sec - start.tv_sec) + finish.tv_usec - start.tv_usec;
    printf("CPU: %lf ms\n", t / 1000);
//CPU serial start

//check
    int errorNum = 0;
    for (int i = 0; i < n; i++) {
        if (host_c[i] != host_d[i]) {
            errorNum ++;
            printf("Error occurs at index: %d: a + b = %d + %d = %d, but c = %d, c2 = %d\n", i, host_a[i], host_b[i], host_a[i] + host_b[i], host_c[i], host_d[i]);
        }
    }
    if (errorNum == 0) {
        printf("Successfully run on GPU and CPU!\n");
    } else {
        printf("%d error(s) occurs!\n", errorNum);
    }

    free(host_a);
    free(host_b);
    free(host_c);
    free(host_d);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}
